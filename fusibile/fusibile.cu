#include "hip/hip_runtime.h"
/* vim: ft=cpp
 * */

//#include <hip/hip_vector_types.h>
#ifdef _WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include "globalstate.h"
#include "algorithmparameters.h"
#include "cameraparameters.h"
#include "linestate.h"
#include "imageinfo.h"
#include "config.h"

#include <hip/hip_vector_types.h> // float4
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "vector_operations.h"
#include "point_cloud_list.h"

#define SAVE_TEXTURE
//#define SMOOTHNESS

#define FORCEINLINE __forceinline__
//#define FORCEINLINE


__device__ float K[16];
__device__ float K_inv[16];

/*__device__ FORCEINLINE __constant__ float4 camerasK[32];*/

/* compute depth value from disparity or disparity value from depth
 * Input:  f         - focal length in pixel
 *         baseline  - baseline between cameras (in meters)
 *         d - either disparity or depth value
 * Output: either depth or disparity value
 */
__device__ FORCEINLINE float disparityDepthConversion_cu ( const float &f, const float &baseline, const float &d ) {
    return f * baseline / d;
}

/* compute depth value from disparity or disparity value from depth
 * Input:  f         - focal length in pixel
 *         baseline  - baseline between cameras (in meters)
 *         d - either disparity or depth value
 * Output: either depth or disparity value
 */
__device__ FORCEINLINE float disparityDepthConversion_cu2 ( const float &f, const Camera_cu &cam_ref, const Camera_cu &cam, const float &d ) {
    float baseline = l2_float4(cam_ref.C4 - cam.C4);
    return f * baseline / d;
}

__device__ FORCEINLINE void get3Dpoint_cu ( float4 * __restrict__ ptX, const Camera_cu &cam, const int2 &p, const float &depth ) {
    // in case camera matrix is not normalized: see page 162, then depth might not be the real depth but w and depth needs to be computed from that first
    const float4 pt = make_float4 (
                                   depth * (float)p.x     - cam.P_col34.x,
                                   depth * (float)p.y     - cam.P_col34.y,
                                   depth                  - cam.P_col34.z,
                                   0);

    matvecmul4 (cam.M_inv, pt, ptX);
}
__device__ FORCEINLINE void get3Dpoint_cu1 ( float4 * __restrict__ ptX, const Camera_cu &cam, const int2 &p) {
    // in case camera matrix is not normalized: see page 162, then depth might not be the real depth but w and depth needs to be computed from that first
    float4 pt;
    pt.x = (float)p.x     - cam.P_col34.x;
    pt.y = (float)p.y     - cam.P_col34.y;
    pt.z = 1.0f           - cam.P_col34.z;

    matvecmul4 (cam.M_inv, pt, ptX);
}
//get d parameter of plane pi = [nT, d]T, which is the distance of the plane to the camera center
__device__ FORCEINLINE float getPlaneDistance_cu ( const float4 &normal, const float4 &X ) {
    return -(dot4(normal,X));
}

__device__ FORCEINLINE void normalize_cu (float4 * __restrict__ v)
{
    const float normSquared = pow2(v->x) + pow2(v->y) + pow2(v->z);
    const float inverse_sqrt = rsqrtf (normSquared);
    v->x *= inverse_sqrt;
    v->y *= inverse_sqrt;
    v->z *= inverse_sqrt;
}
__device__ FORCEINLINE void getViewVector_cu (float4 * __restrict__ v, const Camera_cu &camera, const int2 &p)
{
    get3Dpoint_cu1 (v, camera, p);
    sub((*v), camera.C4);
    normalize_cu(v);
    //v->x=0;
    //v->y=0;
    //v->z=1;
}

__device__ FORCEINLINE float l1_norm(float f) {
    return fabsf(f);
}
__device__ FORCEINLINE float l1_norm(float4 f) {
    return ( fabsf (f.x) +
             fabsf (f.y) +
             fabsf (f.z))*0.3333333f;

}
__device__ FORCEINLINE float l1_norm2(float4 f) {
    return ( fabsf (f.x) +
             fabsf (f.y) +
             fabsf (f.z));

}

/* get angle between two vectors in 3D
 * Input: v1,v2 - vectors
 * Output: angle in radian
 */
__device__ FORCEINLINE float getAngle_cu ( const float4 &v1, const float4 &v2 ) {
    float angle = acosf ( dot4(v1, v2));
    //if angle is not a number the dot product was 1 and thus the two vectors should be identical --> return 0
    if ( angle != angle )
        return 0.0f;
    //if ( acosf ( v1.dot ( v2 ) ) != acosf ( v1.dot ( v2 ) ) )
    //cout << acosf ( v1.dot ( v2 ) ) << " / " << v1.dot ( v2 )<< " / " << v1<< " / " << v2 << endl;
    return angle;
}
__device__ FORCEINLINE void project_on_camera (const float4 &X, const Camera_cu &cam, float2 *pt, float *depth) {
    float4 tmp = make_float4 (0, 0, 0, 0);
    matvecmul4P (cam.P, X, (&tmp));
    pt->x = tmp.x / tmp.z;
    pt->y = tmp.y / tmp.z;
    *depth = tmp.z;
}

/*
 * Simple and fast depth math fusion based on depth map and normal consensus
 */
__global__ void fusibile (GlobalState &gs, int ref_camera)
{
    int2 p = make_int2 ( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y );
    //printf("p is %d %d\n", p.x, p.y);

    const int cols = gs.cameras->cols;
    const int rows = gs.cameras->rows;

    if (p.x>=cols)
        return;
    if (p.y>=rows)
        return;

    const int center = p.y*cols+p.x;

    const CameraParameters_cu &camParams = *(gs.cameras);

    if (gs.lines[ref_camera].used_pixels[center]==1)
        return;

    //printf("ref_camera is %d\n", ref_camera);
    const float4 normal = tex2D<float4> (gs.normals_depths[ref_camera], p.x+0.5f, p.y+0.5f);
    //printf("Normal is %f %f %f\nDepth is %f\n", normal.x, normal.y, normal.z, normal.w);
    /*
     * For each point of the reference camera compute the 3d position corresponding to the corresponding depth.
     * Create a point only if the following conditions are fulfilled:
     * - Projected depths of other cameras does not differ more than gs.params.depthThresh
     * - Angle of normal does not differ more than gs.params.normalThresh
     */
    float depth = normal.w;

    float4 X;
    get3Dpoint_cu (&X, camParams.cameras[ref_camera], p, depth);
    //if (p.x<100 && p.y ==100)
    //printf("3d Point is %f %f %f\n", X.x, X.y, X.z);
    float4 consistent_X = X;
    float4 consistent_normal  = normal;
    float4 consistent_texture4 = tex2D<float4> (gs.imgs[ref_camera], p.x+0.5f, p.y+0.5f);
    int number_consistent = 0;
    //int2 used_list[camParams.viewSelectionSubsetNumber];
    int2 used_list[MAX_IMAGES];
    for ( int i = 0; i < camParams.viewSelectionSubsetNumber; i++ ) {

        int idxCurr = camParams.viewSelectionSubset[i];
        used_list[idxCurr].x=-1;
        used_list[idxCurr].y=-1;
        if (idxCurr == ref_camera)
            continue;

        // Project 3d point X on camera idxCurr
        float2 tmp_pt;
        project_on_camera (X, camParams.cameras[idxCurr], &tmp_pt, &depth);
        //printf("P for camera %d is \n", i);
        //print_matrix (camParams.cameras[idxCurr].P, "camera ");
        //printf("2d point for camera %d is %f %f\n", idxCurr, tmp_pt.x, tmp_pt.y);

        // Boundary check
        if (tmp_pt.x >=0 &&
            tmp_pt.x < cols &&
            tmp_pt.y >=0 &&
            tmp_pt.y < rows) {
            //printf("Boundary check passed\n");

            // Compute interpolated depth and normal for tmp_pt w.r.t. camera ref_camera
            float4 tmp_normal_and_depth; // first 3 components normal, fourth depth
            tmp_normal_and_depth   = tex2D<float4> (gs.normals_depths[idxCurr], tmp_pt.x+0.5f, tmp_pt.y+0.5f);
            //printf("New depth is %f vs %f\n", tmp_normal_and_depth.w, depth);

            const float depth_disp = disparityDepthConversion_cu2                ( camParams.cameras[ref_camera].f, camParams.cameras[ref_camera], camParams.cameras[idxCurr], depth );
            const float tmp_normal_and_depth_disp = disparityDepthConversion_cu2 ( camParams.cameras[ref_camera].f, camParams.cameras[ref_camera], camParams.cameras[idxCurr], tmp_normal_and_depth.w );
            // First consistency check on depth
            if (fabsf(depth_disp - tmp_normal_and_depth_disp) < gs.params->depthThresh) {
                //printf("\tFirst consistency test passed!\n");
                float angle = getAngle_cu (tmp_normal_and_depth, normal); // extract normal
                if (angle < gs.params->normalThresh)
                {
                    //printf("\tSecond consistency test passed!\n");
                    /// All conditions met:
                    //  - average 3d points and normals
                    //  - save resulting point and normal
                    //  - (optional) average texture (not done yet)
                    float4 tmp_X; // 3d point of consistent point on other view
                    int2 tmp_p = make_int2 ((int) tmp_pt.x, (int) tmp_pt.y);

                    get3Dpoint_cu (&tmp_X, camParams.cameras[idxCurr], tmp_p, tmp_normal_and_depth.w);
                    consistent_X      = consistent_X      + tmp_X;
                    //consistent_X      = tmp_X;
                    consistent_normal = consistent_normal + tmp_normal_and_depth;
                    if (gs.params->saveTexture)
                        consistent_texture4 = consistent_texture4 + tex2D<float4> (gs.imgs[idxCurr], tmp_pt.x+0.5f, tmp_pt.y+0.5f);



                    // Save the point for later check
                    //printf ("Saved point on camera %d is %d %d\n", idxCurr, (int)tmp_pt.x, (int)tmp_pt.y);
                    used_list[idxCurr].x=(int)tmp_pt.x;
                    used_list[idxCurr].y=(int)tmp_pt.y;

                    number_consistent++;
                }
            }
        }
        else
            continue;
    }

    // Average normals and points
    consistent_X       = consistent_X       / ((float) number_consistent + 1.0f);
    consistent_normal  = consistent_normal  / ((float) number_consistent + 1.0f);
    consistent_texture4 = consistent_texture4 / ((float) number_consistent + 1.0f);

    // If at least numConsistentThresh point agree:
    // Create point
    // Save normal
    // (optional) save texture
    if (number_consistent >= gs.params->numConsistentThresh) {
        //printf("\tEnough consistent points!\nSaving point %f %f %f", consistent_X.x, consistent_X.y, consistent_X.z);
        if (!gs.params->remove_black_background) // hardcoded for middlebury TODO FIX
        {
            gs.pc->points[center].coord  = consistent_X;
            gs.pc->points[center].normal = consistent_normal;

#ifdef SAVE_TEXTURE
            if (gs.params->saveTexture)
                gs.pc->points[center].texture4 = consistent_texture4;
#endif

//            //// Mark corresponding point on other views as "used"
//            for ( int i = 0; i < camParams.viewSelectionSubsetNumber; i++ ) {
//                int idxCurr = camParams.viewSelectionSubset[i];
//                if (used_list[idxCurr].x==-1)
//                    continue;
//                //printf("Used list point on camera %d is %d %d\n", idxCurr, used_list[idxCurr].x, used_list[idxCurr].y);
//                gs.lines[idxCurr].used_pixels [used_list[idxCurr].x + used_list[idxCurr].y*cols] = 1;
//            }
        }
    }

    return;
}
/* Copy point cloud to global memory */
//template< typename T >
void copy_point_cloud_to_host(GlobalState &gs, int cam, PointCloudList &pc_list)
{
    printf("Processing camera %d\n", cam);
    unsigned int count = pc_list.size;
    for (int y=0; y<gs.pc->rows; y++) {
        for (int x=0; x<gs.pc->cols; x++) {
            Point_cu &p = gs.pc->points[x+y*gs.pc->cols];
            const float4 X      = p.coord;
            const float4 normal = p.normal;
            float texture4[4];
#ifdef SAVE_TEXTURE
            if (gs.params->saveTexture)
            {
                texture4[0] = p.texture4.x;
                texture4[1] = p.texture4.y;
                texture4[2] = p.texture4.z;
                texture4[3] = p.texture4.w;
            }
#endif
            if (count==pc_list.maximum) {
                printf("Not enough space to save points :'(\n... allocating more! :)");
                pc_list.increase_size(pc_list.maximum*2);

            }
            if (X.x != 0 && X.y != 0 && X.z != 0) {
                pc_list.points[count].coord   = X;
                pc_list.points[count].normal  = normal;
#ifdef SAVE_TEXTURE
                pc_list.points[count].texture4[0] = texture4[0];
                pc_list.points[count].texture4[1] = texture4[1];
                pc_list.points[count].texture4[2] = texture4[2];
                pc_list.points[count].texture4[3] = texture4[3];
#endif
                count++;
            }
            p.coord = make_float4(0,0,0,0);
        }
    }
    printf("Found %.2f million points\n", count/1000000.0f);
    pc_list.size = count;
}

template< typename T >
void fusibile_cu(GlobalState &gs, PointCloudList &pc_list, int num_views)
{
#ifdef SHARED
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
#endif

    int rows = gs.cameras->rows;
    int cols = gs.cameras->cols;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Run gipuma\n");
    /*hiprandState* devStates;*/
    //hipMalloc ( &gs.cs, rows*cols*sizeof( hiprandState ) );

    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return ;
    }

    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }
    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA.\n");
        return ;
    }
    //float mind = gs.params.min_disparity;
    //float maxd = gs.params.max_disparity;
    //srand(0);
    //for(int x = 0; x < gs.cameras.cols; x++) {
    //for(int y = 0; y < gs.cameras.rows; y++) {
    //gs.lines.disp[y*gs.cameras.cols+x] = (float)rand()/(float)RAND_MAX * (maxd-mind) + mind;
    //[>printf("%f\n", gs.lines.disp[y*256+x]);<]
    //}
    //}
    /*printf("MAX DISP is %f\n", gs.params.max_disparity);*/
    /*printf("MIN DISP is %f\n", gs.params.min_disparity);*/
    hipSetDevice(i);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*128);
    dim3 grid_size;
    grid_size.x=(cols+BLOCK_W-1)/BLOCK_W;
    grid_size.y=((rows/2)+BLOCK_H-1)/BLOCK_H;
    dim3 block_size;
    block_size.x=BLOCK_W;
    block_size.y=BLOCK_H;

    dim3 grid_size_initrand;
    grid_size_initrand.x=(cols+32-1)/32;
    grid_size_initrand.y=(rows+32-1)/32;
    dim3 block_size_initrand;
    block_size_initrand.x=32;
    block_size_initrand.y=32;

/*     printf("Launching kernel with grid of size %d %d and block of size %d %d and shared size %d %d\nBlock %d %d and radius %d %d and tile %d %d\n",
           grid_size.x,
           grid_size.y,
           block_size.x,
           block_size.y,
           SHARED_SIZE_W,
           SHARED_SIZE_H,
           BLOCK_W,
           BLOCK_H,
           WIN_RADIUS_W,
           WIN_RADIUS_H,
           TILE_W,
           TILE_H
          );
 */    printf("Grid size initrand is grid: %d-%d block: %d-%d\n", grid_size_initrand.x, grid_size_initrand.y, block_size_initrand.x, block_size_initrand.y);

    size_t avail;
    size_t total;
    hipMemGetInfo( &avail, &total );
    size_t used = total - avail;
    printf("Device memory used: %fMB\n", used/1000000.0f);
    printf("Number of iterations is %d\n", gs.params->iterations);
    printf("Blocksize is %dx%d\n", gs.params->box_hsize,gs.params->box_vsize);
    printf("Disparity threshold is \t%f\n", gs.params->depthThresh);
    printf("Normal threshold is \t%f\n", gs.params->normalThresh);
    printf("Number of consistent points is \t%d\n", gs.params->numConsistentThresh);
    printf("Cam scale is \t%f\n", gs.params->cam_scale);

    //int shared_memory_size = sizeof(float)  * SHARED_SIZE ;
    printf("Fusing points\n");
    hipEventRecord(start);

    //printf("Computing final disparity\n");
    //for (int cam=0; cam<10; cam++) {
    for (int cam=0; cam<num_views; cam++) {
        fusibile<<< grid_size_initrand, block_size_initrand, cam>>>(gs, cam);
        hipDeviceSynchronize();
        copy_point_cloud_to_host(gs, cam, pc_list); // slower but saves memory
        hipDeviceSynchronize();
    }

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\t\tELAPSED %f seconds\n", milliseconds/1000.f);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // print results to file
}

int runcuda(GlobalState &gs, PointCloudList &pc_list, int num_views)
{
    printf("Run cuda\n");
    /*GlobalState *gs = new GlobalState;*/
    if(gs.params->color_processing)
        fusibile_cu<float4>(gs, pc_list, num_views);
    else
        fusibile_cu<float>(gs, pc_list, num_views);
    return 0;
}
